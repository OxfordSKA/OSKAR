#include "hip/hip_runtime.h"
/* Copyright (c) 2018-2025, The OSKAR Developers. See LICENSE file. */

#include "convert/define_convert_az_el_to_enu_directions.h"
#include "convert/define_convert_apparent_ra_dec_to_enu_directions.h"
#include "convert/define_convert_cirs_relative_directions_to_enu_directions.h"
#include "convert/define_convert_ecef_to_station_uvw.h"
#include "convert/define_convert_enu_directions_to_az_el.h"
/*#include "convert/define_convert_enu_directions_to_cirs_relative_directions.h"*/
#include "convert/define_convert_enu_directions_to_local_tangent_plane.h"
#include "convert/define_convert_enu_directions_to_relative_directions.h"
#include "convert/define_convert_enu_directions_to_theta_phi.h"
#include "convert/define_convert_lon_lat_to_relative_directions.h"
#include "convert/define_convert_relative_directions_to_enu_directions.h"
#include "convert/define_convert_relative_directions_to_lon_lat.h"
#include "convert/define_convert_station_uvw_to_baseline_uvw.h"
#include "utility/oskar_cuda_registrar.h"
#include "utility/oskar_kernel_macros.h"
#include "utility/oskar_vector_types.h"

/* Kernels */

#define Real float
#define Real2 float2
#define Real4c float4c
#include "convert/src/oskar_convert.cl"
#undef Real
#undef Real2
#undef Real4c

#define Real double
#define Real2 double2
#define Real4c double4c
#include "convert/src/oskar_convert.cl"
#undef Real
#undef Real2
#undef Real4c
